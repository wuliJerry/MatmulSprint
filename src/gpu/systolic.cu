#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 32 // This should match the size of your systolic array

__global__ void systolicMatrixMultiply(int *mat1, int *mat2, int *matRst, size_t M, size_t K, size_t N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
  
    __shared__ int partialResult[THREADS_PER_BLOCK][THREADS_PER_BLOCK];

    for (int k = 0; k < K; ++k) {
        partialResult[threadIdx.y][threadIdx.x] += mat1[row * K + k] * mat2[k * N + col];
        __syncthreads();
    }

    matRst[row * N + col] = partialResult[threadIdx.y][threadIdx.x];
}

int main() {
    int *h_mat1, *h_mat2, *h_matRst;
    int *d_mat1, *d_mat2, *d_matRst;
    size_t M, K, N;
    M = K = N = 4096; // Example size

    // Allocate host memory
    h_mat1 = (int*)malloc(M * K * sizeof(int));
    h_mat2 = (int*)malloc(K * N * sizeof(int));
    h_matRst = (int*)malloc(M * N * sizeof(int));

    // Allocate device memory
    hipMalloc(&d_mat1, M * K * sizeof(int));
    hipMalloc(&d_mat2, K * N * sizeof(int));
    hipMalloc(&d_matRst, M * N * sizeof(int));

    // Initialize host matrices and copy to device
    for (int i = 0; i < M * K; ++i) h_mat1[i] = 1;
    for (int i = 0; i < K * N; ++i) h_mat2[i] = 1;
    hipMemcpy(d_mat1, h_mat1, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, h_mat2, K * N * sizeof(int), hipMemcpyHostToDevice);

    // Call the kernel
    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks(M / threadsPerBlock.x, N / threadsPerBlock.y);
    systolicMatrixMultiply<<<numBlocks, threadsPerBlock>>>(d_mat1, d_mat2, d_matRst, M, K, N);

    // Copy the result back to the host
    hipMemcpy(h_matRst, d_matRst, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_matRst);
    free(h_mat1);
    free(h_mat2);
    free(h_matRst);

    return 0;
}
